#include "hip/hip_runtime.h"
#define GLM_ENABLE_EXPERIMENTAL

#include "physics.cuh"
#include <cmath>
#include <glm/gtx/norm.hpp>
#include <sstream>
#include "hip/hip_runtime_api.h"

__device__ float cudaCalcWpoly6(glm::vec3 i, glm::vec3 j) {    
    float r2 = glm::distance2(i, j);
    return (float)(r2 <= P_H2 ? POLY6_COEFF * std::pow(P_H2 - r2, 3) : 0);
}

__device__ glm::vec3 cudaCalcWspiky(glm::vec3 i, glm::vec3 j) {
    float r = glm::distance(i, j);

    if ((EPS <= r) && (r <= P_H)) {
        float coeff = (float)(-SPIKY_COEFF * std::pow(P_H - r, 2));
        return coeff * glm::normalize(i - j);
    } else {
        return glm::vec3(0.0f);
    }
}

void cudaApplyBodyForces(ParticleSystem *psystem) {
    for (size_t i = 0; i < psystem->num_particles; i++) {
        psystem->vel[i].z -= G * DT;
        psystem->pos[i] = psystem->prevpos[i] + DT * psystem->vel[i];
    }
}

void cudaCalcPartition(ParticleSystem *psystem) {
    Box *box = psystem->box;
    box->clear_partitions();

    for (size_t i = 0; i < psystem->num_particles; i++) {
        box->add_particle(i, psystem->pos[i]);
    }
}

__global__ void cudaKNearestNeighbors(ParticleSystem *psystem) {
    Box *box = psystem->box;

    size_t p = blockIdx.x * blockDim.x + threadIdx.x;
    while (p < psystem->num_particles) {
        glm::vec3 pi = psystem->pos[p];

        // Discretize particle positions into a 3D grid of size P_H
        int x = (int)((pi.x + EPS) / P_H);
        int y = (int)((pi.y + EPS) / P_H);
        int z = (int)((pi.z + EPS) / P_H);

        float dist, max;
        size_t max_idx = 0, num_neighbors = 0;
        glm::vec3 pj;

        for (int i = x - 1; i <= x + 1; i++) {
            for (int j = y - 1; j <= y + 1; j++) {
                for (int k = z - 1; k <= z + 1; k++) {
                    if (i < 0 || j < 0 || k < 0 || (size_t)i >= box->x_partitions ||
                        (size_t)j >= box->y_partitions || (size_t)k >= box->z_partitions) continue;
                    
                    for (size_t l = 0; l < box->get_part_sz(i, j, k); l++) {
                        size_t neighbor = box->get_id_at(i, j, k, l);
                        if (neighbor == p) continue;
                        pj = psystem->pos[neighbor];

                        dist = glm::distance(pi, pj);
                        if (num_neighbors < MAX_NEIGHBORS) {
                            if (dist < P_H) {
                                psystem->neighbors[p + num_neighbors * psystem->num_particles] = neighbor;
                                num_neighbors++;
                            }
                        } else {
                            max = 0.0f;
                            for (size_t idx = 0; idx < num_neighbors; idx++) {
                                size_t neighbor_idx = psystem->neighbors[p + idx * psystem->num_particles];
                                float d = glm::distance(pi, psystem->pos[neighbor_idx]);
                                if (d > max) {
                                    max = d;
                                    max_idx = idx;
                                }
                            }
                            if (dist < max && dist < P_H) {
                                psystem->neighbors[p + max_idx * psystem->num_particles] = neighbor;
                            }
                        }
                    }
                }
            }
        }
        psystem->num_neighbors[p] = num_neighbors;

        p += blockDim.x * gridDim.x;
    }
}

__global__ void cudaCalcLambda(ParticleSystem *psystem) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < psystem->num_particles) {
        glm::vec3 pi = psystem->pos[i];

        glm::vec3 gradPjCi;  // Temporary store for calculated gradients

        // Accumulators for summing over neighbors j
        float rhoI = 0.0f;
        float sumGradPkCi2 = 0.0f;
        glm::vec3 sumGradPiCi = glm::vec3(0.0f);
        glm::vec3 pj;

        for (size_t j = 0; j < psystem->num_neighbors[i]; j++) {
            size_t neighbor_idx = psystem->neighbors[i + j * psystem->num_particles];
            pj = psystem->pos[neighbor_idx];

            rhoI += cudaCalcWpoly6(pi, pj);  // eq (1)
            gradPjCi = cudaCalcWspiky(pi, pj) * RHO_0_INV;  // eq (8)

            sumGradPiCi += gradPjCi;  // eq (9), denominator, k = j
            sumGradPkCi2 += glm::length2(gradPjCi);  // eq (9), k = i
        }
        sumGradPkCi2 += glm::length2(sumGradPiCi);   // eq (9), k = i

        float numerator = rhoI * RHO_0_INV - 1.0f;  // eq (11)
        float denominator = sumGradPkCi2 + RELAXATION_EPS;  // eq (11)

        psystem->lambda[i] = -numerator / denominator;  // eq (11)

        i += blockDim.x * gridDim.x;
    }
}

__global__ void cudaCalcDeltaPos(ParticleSystem *psystem) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < psystem->num_particles) {
        glm::vec3 pi = psystem->pos[i];
        float lambda_i = psystem->lambda[i];

        glm::vec3 dq = pi + DELTA_Q * glm::vec3(1.0f);
        float denom = 1.0f / cudaCalcWpoly6(pi, dq);  // eq (13)

        glm::vec3 dpi = glm::vec3(0.0f);
        glm::vec3 pj;
        float sCorrBase, sCorr, lambda_j;

        for (size_t j = 0; j < psystem->num_neighbors[i]; j++) {
            size_t neighbor_idx = psystem->neighbors[i + j * psystem->num_particles];
            pj = psystem->pos[neighbor_idx];
            lambda_j = psystem->lambda[neighbor_idx];

            sCorrBase = cudaCalcWpoly6(pi, pj) * denom;
            sCorr = -SCORR_K * SCORR_N(sCorrBase);

            dpi += (lambda_i + lambda_j + sCorr) * cudaCalcWspiky(pi, pj);  // eq (14)
        }
        psystem->deltapos[i] = dpi * RHO_0_INV;  // eq (14)

        i += blockDim.x * gridDim.x;
    }
}

__global__ void cudaUpdatePos(ParticleSystem *psystem) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < psystem->num_particles) {
        psystem->pos[i] += psystem->deltapos[i];

        i += blockDim.x * gridDim.x;
    }
}

__global__ void cudaSavePrevPos(ParticleSystem *psystem) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < psystem->num_particles) {
        psystem->prevpos[i] = psystem->pos[i];

        i += blockDim.x * gridDim.x;
    }
}

__global__ void cudaCalcVel(ParticleSystem *psystem) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < psystem->num_particles) {
        psystem->vel[i] = (psystem->pos[i] - psystem->prevpos[i]) / DT;
        psystem->nextvel[i] = psystem->vel[i];

        i += blockDim.x * gridDim.x;
    }
}

__global__ void cudaUpdateVel(ParticleSystem *psystem) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < psystem->num_particles) {
        psystem->vel[i] = psystem->nextvel[i];

        i += blockDim.x * gridDim.x;
    }
}

__global__ void cudaApplyCollisionResponse(ParticleSystem *psystem) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < psystem->num_particles) {
        if (psystem->pos[i].x < SHAKE(psystem->shake_t)) {
            psystem->pos[i].x = 2*SHAKE(psystem->shake_t) -psystem->pos[i].x;
            psystem->vel[i].x *= -1;
        }
        if (psystem->pos[i].x > psystem->box->xybound + SHAKE(psystem->shake_t)) {
            psystem->pos[i].x = 2*(psystem->box->xybound + SHAKE(psystem->shake_t)) - psystem->pos[i].x;
            psystem->vel[i].x *= -1;
        }

        if (psystem->pos[i].y < 0) {
            psystem->pos[i].y *= -1;
            psystem->vel[i].y *= -1;
        }
        if (psystem->pos[i].y > psystem->box->xybound) {
            psystem->pos[i].y = 2*psystem->box->xybound - psystem->pos[i].y;
            psystem->vel[i].x *= -1;
        }

        if (psystem->pos[i].z < 0) {
            psystem->pos[i].z *= -1;
            psystem->vel[i].z *= -1;
        }
        if (psystem->pos[i].z > psystem->box->zbound) {
            psystem->pos[i].z = 2*psystem->box->zbound - psystem->pos[i].z;
            psystem->vel[i].z *= -1;
        }

        i += blockDim.x * gridDim.x;
    }
}

__global__ void cudaCalcVorticityViscosity(ParticleSystem *psystem) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < psystem->num_particles) {
        glm::vec3 pi = psystem->pos[i];
        glm::vec3 vi = psystem->vel[i];

        glm::vec3 vij, pj;
        glm::vec3 wi    = glm::vec3(0.0f);
        glm::vec3 vXSPH = glm::vec3(0.0f);

        for (size_t j = 0; j < psystem->num_neighbors[i]; j++) {
            size_t neighbor_idx = psystem->neighbors[i + j * psystem->num_particles];
            pj = psystem->pos[neighbor_idx];
            vij = psystem->vel[neighbor_idx] - vi;

            wi += glm::cross(vij, cudaCalcWspiky(pi, pj));  // eq (15)
            vXSPH += vij * cudaCalcWpoly6(pi, pj);  // eq (17)
        }
        psystem->nextvel[i] += XSPH_C * vXSPH; // eq (17)
        psystem->vorticity[i] = wi;

        i += blockDim.x * gridDim.x;
    }
}

__global__ void cudaApplyVorticityCorrection(ParticleSystem *psystem) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < psystem->num_particles) {
        glm::vec3 pi = psystem->pos[i];
        glm::vec3 gradwi = glm::vec3(0.0f);
        glm::vec3 pj, wj;

        for (size_t j = 0; j < psystem->num_neighbors[i]; j++) {
            size_t neighbor_idx = psystem->neighbors[i + j * psystem->num_particles];
            pj = psystem->pos[neighbor_idx];
            wj = psystem->vorticity[neighbor_idx];

            gradwi += glm::length(wj) * cudaCalcWspiky(pi, pj);
        }

        /* Avoid normalizing zero values to 1 */
        if (glm::length(gradwi) > EPS) gradwi = glm::normalize(gradwi);

        psystem->nextvel[i] += DT * VORTICITY_EPS * glm::cross(gradwi, psystem->vorticity[i]);

        i += blockDim.x * gridDim.x;
    }
}

void cudaUpdate(ParticleSystem *psystem, ParticleSystem *gpu_psystem, bool shake) {
    unsigned int blocks = 512;
    unsigned int threads_per_block = 512;

    // performed on CPU
    cudaApplyBodyForces(psystem);

    hipMemcpy(gpu_psystem, psystem, offsetof(ParticleSystem, pos), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->pos,           psystem->pos,           psystem->num_particles * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->deltapos,      psystem->deltapos,      psystem->num_particles * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->prevpos,       psystem->prevpos,       psystem->num_particles * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->vel,           psystem->vel,           psystem->num_particles * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->nextvel,       psystem->nextvel,       psystem->num_particles * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->vorticity,     psystem->vorticity,     psystem->num_particles * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->lambda,        psystem->lambda,        psystem->num_particles * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->neighbors,     psystem->neighbors,     psystem->num_particles * MAX_NEIGHBORS * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->num_neighbors, psystem->num_neighbors, psystem->num_particles * sizeof(size_t), hipMemcpyHostToDevice);
    
    // performed on CPU
    cudaCalcPartition(psystem);

    hipMemcpy(gpu_psystem->box, psystem->box, offsetof(Box, partitions), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->box->partitions, psystem->box->partitions,
        psystem->box->total_partitions * psystem->num_particles * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->box->partition_sizes, psystem->box->partition_sizes,
        psystem->box->total_partitions * sizeof(size_t), hipMemcpyHostToDevice);

    cudaKNearestNeighbors<<<blocks, threads_per_block>>>(gpu_psystem);

    for (size_t i = 0; i < SOLVER_ITERATIONS; i++) {
        cudaCalcLambda<<<blocks, threads_per_block>>>(gpu_psystem);
        cudaCalcDeltaPos<<<blocks, threads_per_block>>>(gpu_psystem);
        cudaUpdatePos<<<blocks, threads_per_block>>>(gpu_psystem);
        cudaApplyCollisionResponse<<<blocks, threads_per_block>>>(gpu_psystem);
    }

    cudaCalcVel<<<blocks, threads_per_block>>>(gpu_psystem);
    cudaCalcVorticityViscosity<<<blocks, threads_per_block>>>(gpu_psystem);
    cudaApplyVorticityCorrection<<<blocks, threads_per_block>>>(gpu_psystem);
    cudaUpdateVel<<<blocks, threads_per_block>>>(gpu_psystem);

    cudaSavePrevPos<<<blocks, threads_per_block>>>(gpu_psystem);


    hipMemcpy(psystem->pos,           gpu_psystem->pos,           psystem->num_particles * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->deltapos,      gpu_psystem->deltapos,      psystem->num_particles * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->prevpos,       gpu_psystem->prevpos,       psystem->num_particles * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->vel,           gpu_psystem->vel,           psystem->num_particles * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->nextvel,       gpu_psystem->nextvel,       psystem->num_particles * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->vorticity,     gpu_psystem->vorticity,     psystem->num_particles * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->lambda,        gpu_psystem->lambda,        psystem->num_particles * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->neighbors,     gpu_psystem->neighbors,     psystem->num_particles * MAX_NEIGHBORS * sizeof(size_t), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->num_neighbors, gpu_psystem->num_neighbors, psystem->num_particles * sizeof(size_t), hipMemcpyDeviceToHost);


    psystem->t += DT;
    if (shake) psystem->shake_t += DT;
}



void cudaMallocPsystem(ParticleSystem *psystem, ParticleSystem **gpu_psystem) {
    CUDA_CALL(hipMalloc(gpu_psystem, sizeof(ParticleSystem)));
    //CUDA_CALL(hipMalloc(gpu_psystem, sizeof(ParticleSystem)));
    glm::vec3 *pos;
    CUDA_CALL(hipMalloc(&pos,           psystem->num_particles * sizeof(glm::vec3)));
    CUDA_CALL(hipMemcpy())
    // CUDA_CALL(hipMalloc(&(*gpu_psystem)->deltapos,      psystem->num_particles * sizeof(glm::vec3)));
    // CUDA_CALL(hipMalloc(&(*gpu_psystem)->prevpos,       psystem->num_particles * sizeof(glm::vec3)));
    // CUDA_CALL(hipMalloc(&(*gpu_psystem)->vel,           psystem->num_particles * sizeof(glm::vec3)));
    // CUDA_CALL(hipMalloc(&(*gpu_psystem)->nextvel,       psystem->num_particles * sizeof(glm::vec3)));
    // CUDA_CALL(hipMalloc(&(*gpu_psystem)->vorticity,     psystem->num_particles * sizeof(glm::vec3)));
    // CUDA_CALL(hipMalloc(&(*gpu_psystem)->lambda,        psystem->num_particles * sizeof(float)));
    // CUDA_CALL(hipMalloc(&(*gpu_psystem)->neighbors,     psystem->num_particles * MAX_NEIGHBORS * sizeof(size_t)));
    // CUDA_CALL(hipMalloc(&(*gpu_psystem)->num_neighbors, psystem->num_particles * sizeof(size_t)));

    // CUDA_CALL(hipMalloc(&(*gpu_psystem)->box, sizeof(Box)));
    // CUDA_CALL(hipMalloc(&(*gpu_psystem)->box->partitions,
    //     psystem->box->total_partitions * psystem->num_particles * sizeof(size_t)));
    // CUDA_CALL(hipMalloc(&(*gpu_psystem)->box->partition_sizes,
    //     psystem->box->total_partitions * sizeof(size_t)));
}

void cudaReallocPsystem(ParticleSystem *psystem, ParticleSystem *gpu_psystem) {
    // hipFree(gpu_psystem->pos);
    // hipFree(gpu_psystem->deltapos);
    // hipFree(gpu_psystem->prevpos);
    // hipFree(gpu_psystem->vel);
    // hipFree(gpu_psystem->nextvel);
    // hipFree(gpu_psystem->vorticity);
    // hipFree(gpu_psystem->lambda);
    // hipFree(gpu_psystem->neighbors);
    // hipFree(gpu_psystem->num_neighbors);

    // hipMalloc(&gpu_psystem->pos,           psystem->num_particles * sizeof(glm::vec3));
    // hipMalloc(&gpu_psystem->deltapos,      psystem->num_particles * sizeof(glm::vec3));
    // hipMalloc(&gpu_psystem->prevpos,       psystem->num_particles * sizeof(glm::vec3));
    // hipMalloc(&gpu_psystem->vel,           psystem->num_particles * sizeof(glm::vec3));
    // hipMalloc(&gpu_psystem->nextvel,       psystem->num_particles * sizeof(glm::vec3));
    // hipMalloc(&gpu_psystem->vorticity,     psystem->num_particles * sizeof(glm::vec3));
    // hipMalloc(&gpu_psystem->lambda,        psystem->num_particles * sizeof(float));
    // hipMalloc(&gpu_psystem->neighbors,     psystem->num_particles * MAX_NEIGHBORS * sizeof(size_t));
    // hipMalloc(&gpu_psystem->num_neighbors, psystem->num_particles * sizeof(size_t));

    // hipFree(gpu_psystem->box->partitions);
    
    // hipMalloc(&gpu_psystem->box->partitions,
    //     psystem->box->total_partitions * psystem->num_particles * sizeof(size_t));
}