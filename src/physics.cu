#include "hip/hip_runtime.h"
#define GLM_ENABLE_EXPERIMENTAL

#include "physics.cuh"
#include <cmath>
#include <glm/gtx/norm.hpp>

float calcWpoly6(glm::vec3 i, glm::vec3 j) {    
    float r2 = glm::distance2(i, j);
    return (float)(r2 <= P_H2 ? POLY6_COEFF * std::pow(P_H2 - r2, 3) : 0);
}

glm::vec3 calcWspiky(glm::vec3 i, glm::vec3 j) {
    float r = glm::distance(i, j);

    if ((EPS <= r) && (r <= P_H)) {
        float coeff = (float)(-SPIKY_COEFF * std::pow(P_H - r, 2));
        return coeff * glm::normalize(i - j);
    } else {
        return glm::vec3(0.0f);
    }
}

void applyBodyForces(ParticleSystem *psystem) {
    for (size_t i = 0; i < psystem->num_particles; i++) {
        psystem->vel[i].z -= G * DT;
        psystem->pos[i] = psystem->prevpos[i] + DT * psystem->vel[i];
    }
}

void calcPartition(ParticleSystem *psystem) {
    Box *box = psystem->box;
    box->clear_partitions();

    for (size_t i = 0; i < psystem->num_particles; i++) {
        box->add_particle(i, psystem->pos[i]);
    }
}

__global__ void kNearestNeighbors(ParticleSystem *psystem) {
    Box *box = psystem->box;

    size_t p = blockIdx.x * blockDim.x + threadIdx.x;
    while (p < psystem->num_particles) {
        glm::vec3 pi = psystem->pos[p];

        // Discretize particle positions into a 3D grid of size P_H
        int x = (int)((pi.x + EPS) / P_H);
        int y = (int)((pi.y + EPS) / P_H);
        int z = (int)((pi.z + EPS) / P_H);

        float dist, max;
        size_t max_idx = 0, num_neighbors = 0;
        glm::vec3 pj;

        for (int i = x - 1; i <= x + 1; i++) {
            for (int j = y - 1; j <= y + 1; j++) {
                for (int k = z - 1; k <= z + 1; k++) {
                    if (i < 0 || j < 0 || k < 0 || (size_t)i >= box->x_partitions ||
                        (size_t)j >= box->y_partitions || (size_t)k >= box->z_partitions) continue;
                    
                    for (size_t l = 0; l < box->get_part_sz(i, j, k); l++) {
                        size_t neighbor = box->get_id_at(i, j, k, l);
                        if (neighbor == p) continue;
                        pj = psystem->pos[neighbor];

                        dist = glm::distance(pi, pj);
                        if (num_neighbors < MAX_NEIGHBORS) {
                            if (dist < P_H) {
                                psystem->neighbors[p + num_neighbors * psystem->num_particles] = neighbor;
                                num_neighbors++;
                            }
                        } else {
                            max = 0.0f;
                            for (size_t idx = 0; idx < num_neighbors; idx++) {
                                size_t neighbor_idx = psystem->neighbors[p + idx * psystem->num_particles];
                                float d = glm::distance(pi, psystem->pos[neighbor_idx]);
                                if (d > max) {
                                    max = d;
                                    max_idx = idx;
                                }
                            }
                            if (dist < max && dist < P_H) {
                                psystem->neighbors[p + max_idx * psystem->num_particles] = neighbor;
                            }
                        }
                    }
                }
            }
        }
        psystem->num_neighbors[p] = num_neighbors;

        p += blockDim.x * gridDim.x;
    }
}

__global__ void calcLambda(ParticleSystem *psystem) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < psystem->num_particles) {
        glm::vec3 pi = psystem->pos[i];

        glm::vec3 gradPjCi;  // Temporary store for calculated gradients

        // Accumulators for summing over neighbors j
        float rhoI = 0.0f;
        float sumGradPkCi2 = 0.0f;
        glm::vec3 sumGradPiCi = glm::vec3(0.0f);
        glm::vec3 pj;

        for (size_t j = 0; j < psystem->num_neighbors[i]; j++) {
            size_t neighbor_idx = psystem->neighbors[i + j * psystem->num_particles];
            pj = psystem->pos[neighbor_idx];

            rhoI += calcWpoly6(pi, pj);  // eq (1)
            gradPjCi = calcWspiky(pi, pj) * RHO_0_INV;  // eq (8)

            sumGradPiCi += gradPjCi;  // eq (9), denominator, k = j
            sumGradPkCi2 += glm::length2(gradPjCi);  // eq (9), k = i
        }
        sumGradPkCi2 += glm::length2(sumGradPiCi);   // eq (9), k = i

        float numerator = rhoI * RHO_0_INV - 1.0f;  // eq (11)
        float denominator = sumGradPkCi2 + RELAXATION_EPS;  // eq (11)

        psystem->lambda[i] = -numerator / denominator;  // eq (11)

        i += blockDim.x * gridDim.x;
    }
}

__global__ void calcDeltaPos(ParticleSystem *psystem) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < psystem->num_particles) {
        glm::vec3 pi = psystem->pos[i];
        float lambda_i = psystem->lambda[i];

        glm::vec3 dq = pi + DELTA_Q * glm::vec3(1.0f);
        float denom = 1.0f / calcWpoly6(pi, dq);  // eq (13)

        glm::vec3 dpi = glm::vec3(0.0f);
        glm::vec3 pj;
        float sCorrBase, sCorr, lambda_j;

        for (size_t j = 0; j < psystem->num_neighbors[i]; j++) {
            size_t neighbor_idx = psystem->neighbors[i + j * psystem->num_particles];
            pj = psystem->pos[neighbor_idx];
            lambda_j = psystem->lambda[neighbor_idx];

            sCorrBase = calcWpoly6(pi, pj) * denom;
            sCorr = -SCORR_K * SCORR_N(sCorrBase);

            dpi += (lambda_i + lambda_j + sCorr) * calcWspiky(pi, pj);  // eq (14)
        }
        psystem->deltapos[i] = dpi * RHO_0_INV;  // eq (14)

        i += blockDim.x * gridDim.x;
    }
}

__global__ void updatePos(ParticleSystem *psystem) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < psystem->num_particles) {
        psystem->pos[i] += psystem->deltapos[i];

        i += blockDim.x * gridDim.x;
    }
}

__global__ void savePrevPos(ParticleSystem *psystem) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < psystem->num_particles) {
        psystem->prevpos[i] = psystem->pos[i];

        i += blockDim.x * gridDim.x;
    }
}

__global__ void calcVel(ParticleSystem *psystem) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < psystem->num_particles) {
        psystem->vel[i] = (psystem->pos[i] - psystem->prevpos[i]) / DT;
        psystem->nextvel[i] = psystem->vel[i];

        i += blockDim.x * gridDim.x;
    }
}

__global__ void updateVel(ParticleSystem *psystem) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < psystem->num_particles) {
        psystem->vel[i] = psystem->nextvel[i];

        i += blockDim.x * gridDim.x;
    }
}

__global__ void applyCollisionResponse(ParticleSystem *psystem, size_t xybound, size_t zbound) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < psystem->num_particles) {
        if (psystem->pos[i].x < SHAKE(psystem->shake_t)) {
            psystem->pos[i].x = 2*SHAKE(psystem->shake_t) -psystem->pos[i].x;
            psystem->vel[i].x *= -1;
        }
        if (psystem->pos[i].x > psystem->box->xybound + SHAKE(psystem->shake_t)) {
            psystem->pos[i].x = 2*(psystem->box->xybound + SHAKE(psystem->shake_t)) - psystem->pos[i].x;
            psystem->vel[i].x *= -1;
        }

        if (psystem->pos[i].y < 0) {
            psystem->pos[i].y *= -1;
            psystem->vel[i].y *= -1;
        }
        if (psystem->pos[i].y > psystem->box->xybound) {
            psystem->pos[i].y = 2*psystem->box->xybound - psystem->pos[i].y;
            psystem->vel[i].x *= -1;
        }

        if (psystem->pos[i].z < 0) {
            psystem->pos[i].z *= -1;
            psystem->vel[i].z *= -1;
        }
        if (psystem->pos[i].z > psystem->box->zbound) {
            psystem->pos[i].z = 2*psystem->box->zbound - psystem->pos[i].z;
            psystem->vel[i].z *= -1;
        }

        i += blockDim.x * gridDim.x;
    }
}

__global__ void calcVorticityViscosity(ParticleSystem *psystem) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < psystem->num_particles) {
        glm::vec3 pi = psystem->pos[i];
        glm::vec3 vi = psystem->vel[i];

        glm::vec3 vij, pj;
        glm::vec3 wi    = glm::vec3(0.0f);
        glm::vec3 vXSPH = glm::vec3(0.0f);

        for (size_t j = 0; j < psystem->num_neighbors[i]; j++) {
            size_t neighbor_idx = psystem->neighbors[i + j * psystem->num_particles];
            pj = psystem->pos[neighbor_idx];
            vij = psystem->vel[neighbor_idx] - vi;

            wi += glm::cross(vij, calcWspiky(pi, pj));  // eq (15)
            vXSPH += vij * calcWpoly6(pi, pj);  // eq (17)
        }
        psystem->nextvel[i] += XSPH_C * vXSPH; // eq (17)
        psystem->vorticity[i] = wi;

        i += blockDim.x * gridDim.x;
    }
}

__global__ void applyVorticityCorrection(ParticleSystem *psystem) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < psystem->num_particles) {
        glm::vec3 pi = psystem->pos[i];
        glm::vec3 gradwi = glm::vec3(0.0f);
        glm::vec3 pj, wj;

        for (size_t j = 0; j < psystem->num_neighbors[i]; j++) {
            size_t neighbor_idx = psystem->neighbors[i + j * psystem->num_particles];
            pj = psystem->pos[neighbor_idx];
            wj = psystem->vorticity[neighbor_idx];

            gradwi += glm::length(wj) * calcWspiky(pi, pj);
        }

        /* Avoid normalizing zero values to 1 */
        if (glm::length(gradwi) > EPS) gradwi = glm::normalize(gradwi);

        psystem->nextvel[i] += DT * VORTICITY_EPS * glm::cross(gradwi, psystem->vorticity[i]);

        i += blockDim.x * gridDim.x;
    }
}

void cudaUpdate(ParticleSystem *psystem, ParticleSystem *gpu_psystem, bool shake) {
    // performed on CPU
    applyBodyForces(psystem);

    hipMemcpy(gpu_psystem, psystem, offsetof(ParticleSystem, pos), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->pos,           psystem->pos,           psystem->num_particles * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->deltapos,      psystem->deltapos,      psystem->num_particles * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->prevpos,       psystem->prevpos,       psystem->num_particles * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->vel,           psystem->vel,           psystem->num_particles * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->nextvel,       psystem->nextvel,       psystem->num_particles * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->vorticity,     psystem->vorticity,     psystem->num_particles * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->lambda,        psystem->lambda,        psystem->num_particles * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->neighbors,     psystem->neighbors,     psystem->num_particles * MAX_NEIGHBORS * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->num_neighbors, psystem->num_neighbors, psystem->num_particles * sizeof(size_t), hipMemcpyHostToDevice);
    
    // performed on CPU
    calcPartition(psystem);

    hipMemcpy(gpu_psystem->box, psystem->box, offsetof(Box, partitions), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->box->partitions, psystem->box->partitions,
        psystem->box->total_partitions * psystem->num_particles * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->box->partition_sizes, psystem->box->partition_sizes,
        psystem->box->total_partitions * sizeof(size_t), hipMemcpyHostToDevice);

    kNearestNeighbors<<<blocks, threads_per_block>>>(gpu_psystem);

    for (size_t i = 0; i < SOLVER_ITERATIONS; i++) {
        calcLambda<<<blocks, threads_per_block>>>(gpu_psystem);
        calcDeltaPos<<<blocks, threads_per_block>>>(gpu_psystem);
        updatePos<<<blocks, threads_per_block>>>(gpu_psystem);
        applyCollisionResponse<<<blocks, threads_per_block>>>(gpu_psystem);
    }

    calcVel<<<blocks, threads_per_block>>>(gpu_psystem);
    calcVorticityViscosity<<<blocks, threads_per_block>>>(gpu_psystem);
    applyVorticityCorrection<<<blocks, threads_per_block>>>(gpu_psystem);
    updateVel<<<blocks, threads_per_block>>>(gpu_psystem);

    savePrevPos<<<blocks, threads_per_block>>>(gpu_psystem);


    hipMemcpy(psystem->pos,           gpu_psystem->pos,           psystem->num_particles * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->deltapos,      gpu_psystem->deltapos,      psystem->num_particles * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->prevpos,       gpu_psystem->prevpos,       psystem->num_particles * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->vel,           gpu_psystem->vel,           psystem->num_particles * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->nextvel,       gpu_psystem->nextvel,       psystem->num_particles * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->vorticity,     gpu_psystem->vorticity,     psystem->num_particles * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->lambda,        gpu_psystem->lambda,        psystem->num_particles * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->neighbors,     gpu_psystem->neighbors,     psystem->num_particles * MAX_NEIGHBORS * sizeof(size_t), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->num_neighbors, gpu_psystem->num_neighbors, psystem->num_particles * sizeof(size_t), hipMemcpyDeviceToHost);


    psystem->t += DT;
    if (shake) psystem->shake_t += DT;
}