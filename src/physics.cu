#include "hip/hip_runtime.h"
#define GLM_ENABLE_EXPERIMENTAL

#include "physics.cuh"
#include <cmath>
#include <glm/gtx/norm.hpp>
#include <sstream>
#include "hip/hip_runtime_api.h"

__device__ float cudaCalcWpoly6(glm::vec3 i, glm::vec3 j) {    
    float r2 = glm::distance2(i, j);
    return (float)(r2 <= P_H2 ? POLY6_COEFF * std::pow(P_H2 - r2, 3) : 0);
}

__device__ glm::vec3 cudaCalcWspiky(glm::vec3 i, glm::vec3 j) {
    float r = glm::distance(i, j);

    if ((EPS <= r) && (r <= P_H)) {
        float coeff = (float)(-SPIKY_COEFF * std::pow(P_H - r, 2));
        return coeff * glm::normalize(i - j);
    } else {
        return glm::vec3(0.0f);
    }
}

void cudaApplyBodyForces(ParticleSystem *psystem) {
    for (size_t i = 0; i < psystem->num_particles; i++) {
        psystem->vel[i].z -= G * DT;
        psystem->pos[i] = psystem->prevpos[i] + DT * psystem->vel[i];
    }
}

void cudaCalcPartition(ParticleSystem *psystem) {
    Box *box = psystem->box;
    box->clear_partitions();

    for (size_t i = 0; i < psystem->num_particles; i++) {
        box->add_particle(i, psystem->pos[i]);
    }
}

__global__ void cudaKNearestNeighbors(size_t num_particles, glm::vec3 *pos,
                                      size_t *neighbors, size_t *num_neighbors_arr,
                                      size_t x_partitions, size_t y_partitions,
                                      size_t z_partitions, size_t *partitions,
                                      size_t *partition_sizes
                                      ) {
    size_t p = blockIdx.x * blockDim.x + threadIdx.x;
    while (p < num_particles) {
        glm::vec3 pi = pos[p];

        // Discretize particle positions into a 3D grid of size P_H
        int x = (int)((pi.x + EPS) / P_H);
        int y = (int)((pi.y + EPS) / P_H);
        int z = (int)((pi.z + EPS) / P_H);

        float dist, max;
        size_t max_idx = 0, num_neighbors = 0, part_sz;
        glm::vec3 pj;

        for (int i = x - 1; i <= x + 1; i++) {
            for (int j = y - 1; j <= y + 1; j++) {
                for (int k = z - 1; k <= z + 1; k++) {
                    if (i < 0 || j < 0 || k < 0 || (size_t)i >= x_partitions ||
                        (size_t)j >= y_partitions || (size_t)k >= z_partitions) continue;

                    part_sz = partition_sizes[i * y_partitions * z_partitions + j * z_partitions + k];
                    
                    for (size_t l = 0; l < part_sz; l++) {
                        size_t neighbor = partitions[i * y_partitions *
                                    z_partitions * num_particles +
                                    j * z_partitions * num_particles + 
                                    k * num_particles + l];
                        
                        if (neighbor == p) continue;
                        pj = pos[neighbor];

                        dist = glm::distance(pi, pj);
                        if (num_neighbors < MAX_NEIGHBORS) {
                            if (dist < P_H) {
                                neighbors[p + num_neighbors * num_particles] = neighbor;
                                num_neighbors++;
                            }
                        } else {
                            max = 0.0f;
                            for (size_t idx = 0; idx < num_neighbors; idx++) {
                                size_t neighbor_idx = neighbors[p + idx * num_particles];
                                float d = glm::distance(pi, pos[neighbor_idx]);
                                if (d > max) {
                                    max = d;
                                    max_idx = idx;
                                }
                            }
                            if (dist < max && dist < P_H) {
                                neighbors[p + max_idx * num_particles] = neighbor;
                            }
                        }
                    }
                }
            }
        }
        num_neighbors_arr[p] = num_neighbors;

        p += blockDim.x * gridDim.x;
    }
}

__global__ void cudaCalcLambda(size_t num_particles, glm::vec3 *pos, size_t *neighbors,
                               size_t *num_neighbors, float *lambda) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < num_particles) {
        glm::vec3 pi = pos[i];

        glm::vec3 gradPjCi;  // Temporary store for calculated gradients

        // Accumulators for summing over neighbors j
        float rhoI = 0.0f;
        float sumGradPkCi2 = 0.0f;
        glm::vec3 sumGradPiCi = glm::vec3(0.0f);
        glm::vec3 pj;

        for (size_t j = 0; j < num_neighbors[i]; j++) {
            size_t neighbor_idx = neighbors[i + j * num_particles];
            pj = pos[neighbor_idx];

            rhoI += cudaCalcWpoly6(pi, pj);  // eq (1)
            gradPjCi = cudaCalcWspiky(pi, pj) * RHO_0_INV;  // eq (8)

            sumGradPiCi += gradPjCi;  // eq (9), denominator, k = j
            sumGradPkCi2 += glm::length2(gradPjCi);  // eq (9), k = i
        }
        sumGradPkCi2 += glm::length2(sumGradPiCi);   // eq (9), k = i

        float numerator = rhoI * RHO_0_INV - 1.0f;  // eq (11)
        float denominator = sumGradPkCi2 + RELAXATION_EPS;  // eq (11)

        lambda[i] = -numerator / denominator;  // eq (11)

        i += blockDim.x * gridDim.x;
    }
}

__global__ void cudaCalcDeltaPos(size_t num_particles, glm::vec3 *pos, float *lambda,
                                 size_t *neighbors, size_t *num_neighbors, glm::vec3 *deltapos) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < num_particles) {
        glm::vec3 pi = pos[i];
        float lambda_i = lambda[i];

        glm::vec3 dq = pi + DELTA_Q * glm::vec3(1.0f);
        float denom = 1.0f / cudaCalcWpoly6(pi, dq);  // eq (13)

        glm::vec3 dpi = glm::vec3(0.0f);
        glm::vec3 pj;
        float sCorrBase, sCorr, lambda_j;

        for (size_t j = 0; j < num_neighbors[i]; j++) {
            size_t neighbor_idx = neighbors[i + j * num_particles];
            pj = pos[neighbor_idx];
            lambda_j = lambda[neighbor_idx];

            sCorrBase = cudaCalcWpoly6(pi, pj) * denom;
            sCorr = -SCORR_K * SCORR_N(sCorrBase);

            dpi += (lambda_i + lambda_j + sCorr) * cudaCalcWspiky(pi, pj);  // eq (14)
        }
        deltapos[i] = dpi * RHO_0_INV;  // eq (14)

        i += blockDim.x * gridDim.x;
    }
}

__global__ void cudaUpdatePos(size_t num_particles, glm::vec3 *pos, glm::vec3 *deltapos) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < num_particles) {
        pos[i] += deltapos[i];

        i += blockDim.x * gridDim.x;
    }
}

__global__ void cudaSavePrevPos(size_t num_particles, glm::vec3 *prevpos, glm::vec3 *pos) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < num_particles) {
        prevpos[i] = pos[i];

        i += blockDim.x * gridDim.x;
    }
}

__global__ void cudaCalcVel(size_t num_particles, glm::vec3 *vel, glm::vec3 *pos,
                            glm::vec3 *prevpos, glm::vec3 *nextvel) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < num_particles) {
        vel[i] = (pos[i] - prevpos[i]) / DT;
        nextvel[i] = vel[i];

        i += blockDim.x * gridDim.x;
    }
}

__global__ void cudaUpdateVel(size_t num_particles, glm::vec3 *vel, glm::vec3 *nextvel) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < num_particles) {
        vel[i] = nextvel[i];

        i += blockDim.x * gridDim.x;
    }
}

__global__ void cudaApplyCollisionResponse(size_t num_particles, glm::vec3 *pos,
                                           float shake_t, glm::vec3 *vel,
                                           size_t xybound, size_t zbound) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < num_particles) {
        if (pos[i].x < SHAKE(shake_t)) {
            pos[i].x = 2*SHAKE(shake_t) -pos[i].x;
            vel[i].x *= -1;
        }
        if (pos[i].x > xybound + SHAKE(shake_t)) {
            pos[i].x = 2*(xybound + SHAKE(shake_t)) - pos[i].x;
            vel[i].x *= -1;
        }

        if (pos[i].y < 0) {
            pos[i].y *= -1;
            vel[i].y *= -1;
        }
        if (pos[i].y > xybound) {
            pos[i].y = 2*xybound - pos[i].y;
            vel[i].x *= -1;
        }

        if (pos[i].z < 0) {
            pos[i].z *= -1;
            vel[i].z *= -1;
        }
        if (pos[i].z > zbound) {
            pos[i].z = 2*zbound - pos[i].z;
            vel[i].z *= -1;
        }

        i += blockDim.x * gridDim.x;
    }
}

__global__ void cudaCalcVorticityViscosity(size_t num_particles, glm::vec3 *pos,
                                           glm::vec3 *vel, size_t *neighbors,
                                           size_t *num_neighbors, glm::vec3 *nextvel,
                                           glm::vec3 *vorticity) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < num_particles) {
        glm::vec3 pi = pos[i];
        glm::vec3 vi = vel[i];

        glm::vec3 vij, pj;
        glm::vec3 wi    = glm::vec3(0.0f);
        glm::vec3 vXSPH = glm::vec3(0.0f);

        for (size_t j = 0; j < num_neighbors[i]; j++) {
            size_t neighbor_idx = neighbors[i + j * num_particles];
            pj = pos[neighbor_idx];
            vij = vel[neighbor_idx] - vi;

            wi += glm::cross(vij, cudaCalcWspiky(pi, pj));  // eq (15)
            vXSPH += vij * cudaCalcWpoly6(pi, pj);  // eq (17)
        }
        nextvel[i] += XSPH_C * vXSPH; // eq (17)
        vorticity[i] = wi;

        i += blockDim.x * gridDim.x;
    }
}

__global__ void cudaApplyVorticityCorrection(size_t num_particles, glm::vec3 *pos,
                                             size_t *neighbors, size_t *num_neighbors,
                                             glm::vec3 *vorticity, glm::vec3 *nextvel) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < num_particles) {
        glm::vec3 pi = pos[i];
        glm::vec3 gradwi = glm::vec3(0.0f);
        glm::vec3 pj, wj;

        for (size_t j = 0; j < num_neighbors[i]; j++) {
            size_t neighbor_idx = neighbors[i + j * num_particles];
            pj = pos[neighbor_idx];
            wj = vorticity[neighbor_idx];

            gradwi += glm::length(wj) * cudaCalcWspiky(pi, pj);
        }

        /* Avoid normalizing zero values to 1 */
        if (glm::length(gradwi) > EPS) gradwi = glm::normalize(gradwi);

        nextvel[i] += DT * VORTICITY_EPS * glm::cross(gradwi, vorticity[i]);

        i += blockDim.x * gridDim.x;
    }
}

void cudaUpdate(ParticleSystem *psystem, ParticleSystem *gpu_psystem, bool shake) {
    // performed on CPU
    cudaApplyBodyForces(psystem);

    hipMemcpy(gpu_psystem->pos,           psystem->pos,           psystem->num_particles * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->deltapos,      psystem->deltapos,      psystem->num_particles * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->prevpos,       psystem->prevpos,       psystem->num_particles * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->vel,           psystem->vel,           psystem->num_particles * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->nextvel,       psystem->nextvel,       psystem->num_particles * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->vorticity,     psystem->vorticity,     psystem->num_particles * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->lambda,        psystem->lambda,        psystem->num_particles * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->neighbors,     psystem->neighbors,     psystem->num_particles * MAX_NEIGHBORS * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->num_neighbors, psystem->num_neighbors, psystem->num_particles * sizeof(size_t), hipMemcpyHostToDevice);

    // performed on CPU to avoid race conditions
    cudaCalcPartition(psystem);

    hipMemcpy(gpu_psystem->box->partitions, psystem->box->partitions,
        psystem->box->total_partitions * psystem->num_particles * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(gpu_psystem->box->partition_sizes, psystem->box->partition_sizes,
        psystem->box->total_partitions * sizeof(size_t), hipMemcpyHostToDevice);

    cudaKNearestNeighbors<<<BLOCKS, THREADS_PER_BLOCK>>>(psystem->num_particles,
                                                         gpu_psystem->pos,
                                                         gpu_psystem->neighbors,
                                                         gpu_psystem->num_neighbors,
                                                         psystem->box->x_partitions,
                                                         psystem->box->y_partitions,
                                                         psystem->box->z_partitions,
                                                         gpu_psystem->box->partitions,
                                                         gpu_psystem->box->partition_sizes
                                                         );

    for (size_t i = 0; i < SOLVER_ITERATIONS; i++) {
        cudaCalcLambda<<<BLOCKS, THREADS_PER_BLOCK>>>(psystem->num_particles,
                                                      gpu_psystem->pos,
                                                      gpu_psystem->neighbors,
                                                      gpu_psystem->num_neighbors,
                                                      gpu_psystem->lambda);

        cudaCalcDeltaPos<<<BLOCKS, THREADS_PER_BLOCK>>>(psystem->num_particles,
                                                        gpu_psystem->pos,
                                                        gpu_psystem->lambda,
                                                        gpu_psystem->neighbors,
                                                        gpu_psystem->num_neighbors,
                                                        gpu_psystem->deltapos);

        cudaUpdatePos<<<BLOCKS, THREADS_PER_BLOCK>>>(psystem->num_particles,
                                                     gpu_psystem->pos,
                                                     gpu_psystem->deltapos);

        cudaApplyCollisionResponse<<<BLOCKS, THREADS_PER_BLOCK>>>(psystem->num_particles,
                                                                  gpu_psystem->pos,
                                                                  psystem->shake_t,
                                                                  gpu_psystem->vel,
                                                                  psystem->box->xybound,
                                                                  psystem->box->zbound);
    }

    cudaCalcVel<<<BLOCKS, THREADS_PER_BLOCK>>>(psystem->num_particles,
                                               gpu_psystem->vel,
                                               gpu_psystem->pos,
                                               gpu_psystem->prevpos,
                                               gpu_psystem->nextvel);

    cudaCalcVorticityViscosity<<<BLOCKS, THREADS_PER_BLOCK>>>(psystem->num_particles,
                                                              gpu_psystem->pos,
                                                              gpu_psystem->vel,
                                                              gpu_psystem->neighbors,
                                                              gpu_psystem->num_neighbors,
                                                              gpu_psystem->nextvel,
                                                              gpu_psystem->vorticity);
    
    cudaApplyVorticityCorrection<<<BLOCKS, THREADS_PER_BLOCK>>>(psystem->num_particles,
                                                                gpu_psystem->pos,
                                                                gpu_psystem->neighbors,
                                                                gpu_psystem->num_neighbors,
                                                                gpu_psystem->vorticity,
                                                                gpu_psystem->nextvel);
    
    cudaUpdateVel<<<BLOCKS, THREADS_PER_BLOCK>>>(psystem->num_particles,
                                                 gpu_psystem->vel,
                                                 gpu_psystem->nextvel);

    cudaSavePrevPos<<<BLOCKS, THREADS_PER_BLOCK>>>(psystem->num_particles,
                                                   gpu_psystem->prevpos,
                                                   gpu_psystem->pos);
    

    hipMemcpy(psystem->pos,           gpu_psystem->pos,           psystem->num_particles * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->deltapos,      gpu_psystem->deltapos,      psystem->num_particles * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->prevpos,       gpu_psystem->prevpos,       psystem->num_particles * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->vel,           gpu_psystem->vel,           psystem->num_particles * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->nextvel,       gpu_psystem->nextvel,       psystem->num_particles * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->vorticity,     gpu_psystem->vorticity,     psystem->num_particles * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->lambda,        gpu_psystem->lambda,        psystem->num_particles * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->neighbors,     gpu_psystem->neighbors,     psystem->num_particles * MAX_NEIGHBORS * sizeof(size_t), hipMemcpyDeviceToHost);
    hipMemcpy(psystem->num_neighbors, gpu_psystem->num_neighbors, psystem->num_particles * sizeof(size_t), hipMemcpyDeviceToHost);


    psystem->t += DT;
    if (shake) psystem->shake_t += DT;
}


void cudaMallocPsystem(ParticleSystem *psystem, ParticleSystem *gpu_psystem) {
    glm::vec3 *pos;
    CUDA_CALL(hipMalloc(&pos, psystem->num_particles * sizeof(glm::vec3)));
    gpu_psystem->pos = pos;

    glm::vec3 *deltapos;
    CUDA_CALL(hipMalloc(&deltapos, psystem->num_particles * sizeof(glm::vec3)));
    gpu_psystem->deltapos = deltapos;

    glm::vec3 *prevpos;
    CUDA_CALL(hipMalloc(&prevpos, psystem->num_particles * sizeof(glm::vec3)));
    gpu_psystem->prevpos = prevpos;

    glm::vec3 *vel;
    CUDA_CALL(hipMalloc(&vel, psystem->num_particles * sizeof(glm::vec3)));
    gpu_psystem->vel = vel;

    glm::vec3 *nextvel;
    CUDA_CALL(hipMalloc(&nextvel, psystem->num_particles * sizeof(glm::vec3)));
    gpu_psystem->nextvel = nextvel;

    glm::vec3 *vorticity;
    CUDA_CALL(hipMalloc(&vorticity, psystem->num_particles * sizeof(glm::vec3)));
    gpu_psystem->vorticity = vorticity;

    float *lambda;
    CUDA_CALL(hipMalloc(&lambda, psystem->num_particles * sizeof(float)));
    gpu_psystem->lambda = lambda;
    
    size_t *neighbors;
    CUDA_CALL(hipMalloc(&neighbors, psystem->num_particles * MAX_NEIGHBORS * sizeof(size_t)));
    gpu_psystem->neighbors = neighbors;

    size_t *num_neighbors;
    CUDA_CALL(hipMalloc(&num_neighbors, psystem->num_particles * sizeof(size_t)));
    gpu_psystem->num_neighbors = num_neighbors;

    size_t *partitions;
    CUDA_CALL(hipMalloc(&partitions, psystem->box->total_partitions * psystem->num_particles * sizeof(size_t)));
    gpu_psystem->box->partitions = partitions;

    size_t *partition_sizes;
    CUDA_CALL(hipMalloc(&partition_sizes, psystem->box->total_partitions * sizeof(size_t)));
    gpu_psystem->box->partition_sizes = partition_sizes;
}

void cudaReallocPsystem(ParticleSystem *psystem, ParticleSystem *gpu_psystem) {
    hipFree(gpu_psystem->pos);
    hipFree(gpu_psystem->deltapos);
    hipFree(gpu_psystem->prevpos);
    hipFree(gpu_psystem->vel);
    hipFree(gpu_psystem->nextvel);
    hipFree(gpu_psystem->vorticity);
    hipFree(gpu_psystem->lambda);
    hipFree(gpu_psystem->neighbors);
    hipFree(gpu_psystem->num_neighbors);
    hipFree(gpu_psystem->box->partitions);
    hipFree(gpu_psystem->box->partition_sizes);

    cudaMallocPsystem(psystem, gpu_psystem);
}